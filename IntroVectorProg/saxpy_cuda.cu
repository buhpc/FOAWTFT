
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdlib>
#include <iostream>

// clang-format off
#ifndef N
#  define N 20
#endif

#ifndef ITRS
#  define ITRS 1
#endif
// clang-format on

__global__ void saxpy_cuda(size_t n, float a, float *x, float *y, float *z) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < n)
    z[idx] = (a * x[idx]) + y[idx];
}

int main(int argc, const char *argv[]) {
  using namespace std::chrono;
  size_t n;
  int itrs;
  if (argc < 3) {
    n = 1 << N;
    itrs = ITRS;
  } else {
    n = 1 << std::atoi(argv[1]);
    itrs = std::atoi(argv[2]);
  }

  // we need to use posix_memalign API here
  // since operator new does not guaratee alignment
  float a = 8.0f;
  float *x = static_cast<float *>(aligned_alloc(64, n * sizeof(float)));
  float *y = static_cast<float *>(aligned_alloc(64, n * sizeof(float)));
  float *z = static_cast<float *>(aligned_alloc(64, n * sizeof(float)));
  for (size_t i = 0; i < n; i++) {
    x[i] = 2.0f;
    y[i] = 2.0f;
    z[i] = 0.0f;
  }

  float *d_x;
  float *d_y;
  float *d_z;
  hipMalloc(&d_x, n * sizeof(float));
  hipMalloc(&d_y, n * sizeof(float));
  hipMalloc(&d_z, n * sizeof(float));
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, n * sizeof(float), hipMemcpyHostToDevice);

  auto start = high_resolution_clock::now();
  for (auto i = 0; i < itrs; i++) {
    saxpy_cuda<<<(n + 255) / 256, 256>>>(n, a, d_x, d_y, d_z);
  }
  auto end = high_resolution_clock::now();

  auto diff = duration_cast<microseconds>(end - start);
  std::cout << "CUDA vectorized saxpy:\n"
            << "\tIterations = " << itrs << "\n\t"
            << "Average time = " << diff.count() / itrs
            << " micros. Total time= " << diff.count() << " micros.\n";
  return 0;
}